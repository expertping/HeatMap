#include "hip/hip_runtime.h"
#ifndef GENERATE_KERNEL_H
#define GENERATE_KERNEL_H 1

#define E 2.718281828459          
#define SQRT2PI 2.5066282746

//����ɫ��洢����ӳ�䵽һ�������ϣ��Ϳ������������ʽ���д�ȡ
texture<uchar4, 2, hipReadModeNormalizedFloat> colorTex;

//������ĳ��ԴӰ���µ�ǰ��ĸ�˹ֵ
__device__
float getGaussian(float4* sourcePoint,float trgLon,float trgLat)
{
	//�����ر�ʾ��λ�ú��ȵ�����λ�õľ���
	float spaceLon = trgLon - (*sourcePoint).z;
	float spaceLat = trgLat - (*sourcePoint).w;
	float pointSpace2 = spaceLon*spaceLon + spaceLat*spaceLat;
	
	//��˹��ʽ�ķ�ĸ
	float denominator = (*sourcePoint).y * SQRT2PI;
	//��˹��ʽ�ķ���
	
	float numerator = pow(E,(pointSpace2/(-2.0*(*sourcePoint).y*(*sourcePoint).y))); 
	
	//���Ҫ���Կ���ϵ��
	return (*sourcePoint).x*numerator/denominator;
}

//������������
__device__
float dis_PP(float x1,float y1,float x2,float y2)
{
    return sqrt( (x1-x2)*(x1-x2) + (y1-y2)*(y1-y2) );
}

//�����̺߳˺������˺������������ã�������һ�����ص�����ֵ

__global__ 
void generateTexKernel(float4* targetBuffer,unsigned int trgPitch,float4* sourceBuffer,float4* treeBuffer,unsigned int pointNum,unsigned int imageWidth, unsigned int imageHeight,float minLon,float maxLon,float minLat,float maxLat,int layer)
{
	//���㵱ǰ�߳��������̷߳�����������λ�ã����ھ���������������λ��
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
	//����̺߳ų����������С�Ͳ��ü��㣬ֱ���˳�����
	if( x < imageWidth && y < imageHeight )
    {
		//�����̺߳ż��㵽��ǰ���ص�λ��
		float4* target = (float4*)(((char*) targetBuffer) + trgPitch * y ) + x;
		
		//if( (*target).w < 0.9999999999 )
		{
			//���㵱ǰ���ر�ʾ��ʵ������
			float lon = ((float)(x)/(float)imageWidth)*(float)(maxLon - minLon)+(float)minLon;
			float lat = ((float)(y)/(float)imageHeight)*(float)(maxLat - minLat)+(float)minLat;

			float num = (*target).w;//��ǰ���ر�ʾ��ʵ��������ȶ�ֵ

			int treenum = 0;//��ǰ���ʵ��Ĳ��������±�
			for(int i=0;i<layer;i++)//�����Ĳ��������У���ǰ���ر�ʾ�ĵ�������ֻ�������ڵ�ǰ���ڵ���ĸ��ӽڵ��ʾ�ĵ���Χ�е�һ��,���жϵĴ����������Ĳ���
			{

				int maxindex = int((*(treeBuffer+treenum)).z);//��Ҫ������ȵ����ȵ������е�����±�
				int minindex = int((*(treeBuffer+treenum)).w);//��Ҫ������ȵ����ȵ������е���С�±�
			    float midlon = (*(treeBuffer+treenum)).x;//��ǰ���ڵ����ĵ���Χ�ľ�����ֵ
				float midlat = (*(treeBuffer+treenum)).y;//��ǰ���ڵ����ĵ���Χ��γ����ֵ

				if(minindex!=-1)//�ڵ�ǰ���ڵ��ʾ�ĵ���Χ�����ȵ�
				{

					for(int j=minindex;j<=maxindex;j++)//����÷�Χ�ڵ��ȵ�������ȶ�ֵ�������ۼ�
				  {
                      float dis = dis_PP(lon,lat,(*(sourceBuffer+j)).z,(*(sourceBuffer+j)).w);
                      float s = 3 * ((*(sourceBuffer+j)).y);
                     if (dis<=s)
                     {
                         num += getGaussian(sourceBuffer+j,lon,lat)*(0.9-num);
                     }
					 

				  }
				}
				//�����������жϵ�ǰ���ر�ʾ�ĵ����������ڵ�ǰ���ڵ���ĸ��ӽڵ��е���һ��
				if(lon<midlon && lat<midlat)
				{
					treenum = treenum*4+1;
				}
				else 
					if(lon>=midlon && lat<midlat)
					{
						treenum = treenum*4+2;
					}
					else
						if(lon<midlon && lat>=midlat)
						{
							treenum = treenum*4+3;
						}
						else
						{
							treenum = treenum*4+4;
						}

				
			}

			////��˳����������ȵ�Ե�ǰ���ص�Ӱ�죬��������Ӱ��ֵ����
			//for(unsigned int i=0;i<pointNum;i++)
			//{			
			//	num += getGaussian(sourceBuffer+i,lon,lat)*(0.9-num);
			//}	

		//	float _x = num>1.0?1.0:num;
			//ȥ�󶨵���ɫ�������ȡ��ɫ	
		//	float4 color = tex2D( colorTex, _x*1.111111, 0.5 );

			//�����͸������ʱֻ��˴����ȶ�ǿ�����
		//	color.w = _x;
			(*target) = make_float4(num,num,num,num);
		}
	}
}


extern "C"
void generateTexFromHeatPoint(const dim3& blocks,
							  const dim3& threads,
							  void* texBuffer,
							  unsigned int texPitch,
							  void* heatPoints,
							  void* treePoints,
							  void* colorMap,
							  unsigned int pointNum,
							  unsigned int imageWidth,
							  unsigned int imageHeight,
							  float minLon,
							  float maxLon,
							  float minLat,
							  float maxLat,
							  int times,
							  int tnum,
							  int tlayer)
{
	colorTex.normalized = true;                      // normalized texture coordinates (element of [0:1])
    colorTex.filterMode = hipFilterModeLinear;      // bilinear interpolation 
    colorTex.addressMode[0] = hipAddressModeClamp;  // wrap texture coordinates
    colorTex.addressMode[1] = hipAddressModeClamp;

    // ���Դ�����󶨵�һ������
    hipBindTextureToArray( colorTex, reinterpret_cast<hipArray*>(colorMap) );

	generateTexKernel<<<blocks,threads>>>(reinterpret_cast<float4*>(texBuffer),
		texPitch,
		reinterpret_cast<float4*>(heatPoints),
		reinterpret_cast<float4*>(treePoints),
		pointNum,
		imageWidth,
		imageHeight,
		minLon,
		maxLon,
		minLat,
		maxLat,
		tlayer);
		hipDeviceSynchronize();

	
	

	
	//hipMemcpy2D(texBuffer,texPitch,tmpTexBuffer,texPitch,texPitch,imageHeight,hipMemcpyDeviceToDevice);


}

#endif